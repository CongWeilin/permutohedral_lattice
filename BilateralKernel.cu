
#include <hip/hip_runtime.h>
// kernel_example.cu.cc
//#define GOOGLE_CUDA 1

#ifdef GOOGLE_CUDA
#define EIGEN_USE_GPU
#include "BilateralKernel.h"

#include "tensorflow/core/framework/op_kernel.h"
//#include "tensorflow/core/util/cuda_kernel_helper.h"
#include "PermutohedralLatticeGPU.cu"

using namespace tensorflow;

using GPUDevice = Eigen::GpuDevice;


// Define the GPU implementation that launches the CUDA kernel.
template <typename T>
void ExampleFunctor<GPUDevice, T>::operator()(const GPUDevice& d,
                                              T* output,
                                              const T *input,
                                              const T *reference_image,
                                              int num_super_pixels,
                                              int n_spatial_dims,
                                              int *spatial_dims,
                                              int n_input_channels,
                                              int n_reference_channels,
                                              float theta_alpha,
                                              float theta_beta,
                                              bool reverse) {

    int pd = n_reference_channels + n_spatial_dims;
    int vd = n_input_channels + 1;
    int n = num_super_pixels;
    //
    int* spatial_dims_gpu;
    cudaMalloc((void**)&(spatial_dims_gpu), n_spatial_dims*sizeof(int));
    cudaMemcpy(spatial_dims_gpu, spatial_dims, n_spatial_dims*sizeof(int), cudaMemcpyHostToDevice);


    T* positions;
    cudaMalloc((void**)&(positions), n*pd*sizeof(T));

    printf("%d %d %d %f %f\n", n_reference_channels, num_super_pixels, n_spatial_dims, theta_alpha, theta_beta);
    for(int i=0; i < n_spatial_dims; i++)
        printf("%d", spatial_dims[i]);

    compute_bilateral_kernel_gpu(reference_image,
                                 positions,
                                 num_super_pixels,
                                 n_reference_channels,
                                 n_spatial_dims,
                                 spatial_dims_gpu,
                                 theta_alpha,
                                 theta_beta);


    lattice_filter_gpu(output, input, positions, pd, vd, n, reverse);
    cudaFree(positions);
    cudaFree(spatial_dims_gpu);
}

// Explicitly instantiate functors for the types of OpKernels registered.
template struct ExampleFunctor<GPUDevice, float>;
//template struct ExampleFunctor<GPUDevice, int32>;

#endif  // GOOGLE_CUDA